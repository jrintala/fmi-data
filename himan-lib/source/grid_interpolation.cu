#include "hip/hip_runtime.h"
#include "cuda_plugin_helper.h"
#include "grid_interpolation.h"
#include <cusp/array1d.h>
#include <cusp/array2d.h>
#include <cusp/multiply.h>
#include <cusp/transpose.h>

#include <cusp/lapack/lapack.h>

void himan::grid_interpolation::interpolate(std::vector<double>& target,const std::vector<double>& source, const himan::grid_interpolation::interpolation_kernel& kernel)
{
        cusp::array1d<double,cusp::host_memory> target_vector = target;
        cusp::array1d<double,cusp::host_memory> source_vector = source;

        cusp::multiply(kernel.kernel, source_vector, target_vector);

        std::copy(target_vector.begin(),target_vector.end(),target.begin());
}

std::vector<double> himan::grid_interpolation::get_weights(const himan::point& a, const himan::point& b, const himan::point& c, const himan::point& d, const himan::point& p)
{
		if (p==a) return std::vector<double> {1.0, 0.0, 0.0, 0.0};
                if (p==b) return std::vector<double> {0.0, 1.0, 0.0, 0.0};
                if (p==c) return std::vector<double> {0.0, 0.0, 1.0, 0.0};
                if (p==d) return std::vector<double> {1.0, 0.0, 0.0, 1.0};

		std::vector<double> ret(4);

#ifdef DEBUG
		himan::grid_interpolation::check_geometry(a,b,c,d,p);
#endif

		// points form rectangle
		if (a.X() == c.X() && a.Y() == b.Y() && d.X() == b.X() && d.Y() == c.Y())
		{
			/*
			 * Compute a linear map from points c,b to unit square (0|0), (1|1)
			 * by solving the linear equations:
			 *
			 * xi = C0 + C1*x
			 * yi = C2 + C3*y
			 *
			 * where xi,yi denotes coordinates in unit square and x,y in latlon space.
			 *
			 * In matrix-vector form this can be written as
                         *
			 *                                          |    1     1     0     0|
			 * |xi_0 xi_1 yi_0 yi_1| =  |C0 C1 C2 C3| * |   x0    x1     0     0|
			 *                                          |    0     0     1     1|
			 *                                          |    0     0    y0    y1|
                         * with C0, C1, C2, C3 to solve for. In the following code segment this is done through manually inverting the 4x4 matrix first and then 
			 * compute coefficentes by matrix-vector multiplication.
			 */

			// Matrices and Vectors
                	cusp::array2d<double,cusp::host_memory> A(4,4,0.0);
                	cusp::array1d<double,cusp::host_memory> xi(4);
                        cusp::array1d<double,cusp::host_memory> C(4);
                        cusp::array2d<double,cusp::host_memory> x(2,4,0.0);
                        cusp::array1d<double,cusp::host_memory> phi(2);

			// This is the inverted 4x4 matrix
                	A(0,0) = b.X()/(b.X()-c.X());
                	A(0,1) = -c.X()/(b.X()-c.X());
                	A(1,0) = -1/(b.X()-c.X());
               		A(1,1) = 1/(b.X()-c.X());

                	A(2,2) = b.Y()/(b.Y()-c.Y());
                	A(2,3) = -c.Y()/(b.Y()-c.Y());
                	A(3,2) = -1/(b.Y()-c.Y());
                	A(3,3) =  1/(b.Y()-c.Y());

			// Corner coordinates of unit square
			xi[0] = 0;
			xi[1] = 1;
			xi[2] = 0;
			xi[3] = 1;

			// Compute coefficients
                	cusp::multiply(A,xi,C);

			/*
                         * Apply coordinate transformation to transform point p from latlon space to point phi in unit square.
                         *
                         *                               |  1   0|
			 * |xi_p yi_p| = |C0 C1 C2 C3| * |x_p   0|
			 *                               |  0   1|
                         *                               |  0 y_p|
			 */

			x(0,0) = 1; x(0,1) = p.X();
			x(1,2) = 1; x(1,3) = p.Y();

			cusp::multiply(x,C,phi);

			/*
			 * Calculate interpolation weights using bilinear interpolation formulae
			 */

			ret[0] = (1-phi[0])*phi[1];
			ret[1] = phi[0]*phi[1];
			ret[2] = (1-phi[0])*(1-phi[1]);
			ret[3] = phi[0]*(1-phi[1]);

			return ret;
		}
		// points form generic convex quadrilateral
		else
		{
                        /*
                         * Compute a linear map from points a,b,c,d to unit square (0|1), (1|1), (0|0) , (1|0)
                         * by solving the linear equations:
                         *
                         * xi = C0 + C1*x + C2*y + C3*x*y
                         * yi = C4 + C5*x + C6*y + C7*x*y
			 *
                         * where xi,yi denotes coordinates in unit square and x,y in latlon space.
                         *
                         * In matrix-vector form this can be written as
                         *
                         *                                          |    1     1     1     1|
                         * |xi_0 xi_1 xi_2 xi_3| =  |C0 C1 C2 C3| * |   x0    x1    x2    x3|
                         * |yi_0 yi_1 yi_2 yi_3|    |C4 C5 C6 C7|   |   y0    y1    y2    y3|
                         *                                          |x0*y0 x1*y1 x2*y2 x3*y3|
			 *
			 */

			// Matrices and Vectors
			cusp::array2d<double,cusp::host_memory> A(4,4);
			cusp::array2d<double,cusp::host_memory> xi(4,2);
                        cusp::array2d<double,cusp::host_memory> xi_trans(2,4);
                        cusp::array1d<int,cusp::host_memory> piv;
                        cusp::array1d<double,cusp::host_memory> x(4);
                        cusp::array1d<double,cusp::host_memory> phi(2);

			// Construct linear system of equations to be solved
			A(0,0) = 1;           A(1,0) = 1;           A(2,0) = 1;           A(3,0) = 1;
			A(0,1) = a.X();       A(1,1) = b.X();       A(2,1) = c.X();       A(3,1) = d.X();
			A(0,2) = a.Y();       A(1,2) = b.Y();       A(2,2) = c.Y();       A(3,2) = d.Y();
			A(0,3) = a.X()*a.Y(); A(1,3) = b.X()*b.Y(); A(2,3) = c.X()*c.Y(); A(3,3) = d.X()*d.Y();
			
                        xi(0,0) = 0; xi(1,0) = 1; xi(2,0) = 0; xi(3,0) = 1;
                        xi(0,1) = 1; xi(1,1) = 1; xi(2,1) = 0; xi(3,1) = 0;

			// Solve linear system
			cusp::lapack::gesv(A,xi,piv);

                        x[0] = 1; x[1] = p.X(); x[2] = p.Y(); x[3] = p.X()*p.Y();

			cusp::transpose(xi,xi_trans);
			cusp::multiply(xi_trans,x,phi);

                        ret[0] = (1-phi[0])*phi[1];
                        ret[1] = phi[0]*phi[1];
                        ret[2] = (1-phi[0])*(1-phi[1]);
                        ret[3] = phi[0]*(1-phi[1]);

			return ret;
		}
}
